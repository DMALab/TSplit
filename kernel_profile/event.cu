    // Insert the begin and end event.
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventRecord(start,0);

    float elapsedTime;
    hipEventCreate(&stop);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start,stop);
    p->time = elapsedTime;