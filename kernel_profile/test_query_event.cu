#include<hip/hip_runtime.h>
#include<cstdio>
#include<iostream>

int main(){
    hipStream_t cpu2gpu, gpu2cpu;
    hipStreamCreate(&cpu2gpu);
    hipStreamCreate(&gpu2cpu);

    hipEvent_t cpu2gpu_event, gpu2cpu_event;
    hipEventCreate(&cpu2gpu_event);
    hipEventCreate(&gpu2cpu_event);

    int size = 1000 * 1000;
    void *dev_ptr;
    void *host_ptr;

    hipMalloc(&dev_ptr, size);
    hipHostMalloc(&host_ptr, size, hipHostMallocDefault);

    hipMemcpyAsync(dev_ptr, host_ptr, size, hipMemcpyHostToDevice, gpu2cpu);
    hipEventRecord(gpu2cpu_event, gpu2cpu);

    hipError_t flags = hipErrorNotReady;
    while(flags == hipErrorNotReady){
        flags = hipEventQuery(gpu2cpu_event);
        if (flags == hipErrorNotReady){
            std::cout<<"hipErrorNotReady"<<std::endl;
        }
        else if (flags == hipSuccess){
            std::cout<<"hipSuccess"<<std::endl;
        }
        else{
            std::cout<<"Error"<<std::endl;
        }
    }

    return 0;

}