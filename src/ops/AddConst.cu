#include "hip/hip_runtime.h"
#include "gpu_runtime.h"

__global__ void add_const_kernel(const float *input, float *output, float value, size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size) return;
    output[ind] = input[ind] + value;
  }
  
int DLGpuMatrixElementwiseAddByConst(const DLArrayHandle input, float val,
                                       DLArrayHandle output, DLStreamHandle stream_handle = NULL, ProfilerHandle p = NULL){
                                           
    if(p != NULL){
        int size_input = 1, size_output = 1;
        for(int i = 0; i < input -> ndim; i++)
            size_input *= input -> shape[i];
        for(int i = 0; i < output -> ndim; i++)
            size_output *= output -> shape[i];
        p -> input_memory = 1.0 * (size_input) * sizeof(float) / 1024 / 1024;
        p -> output_memory = 1.0 * size_output * sizeof(float) / 1024 / 1024;
        p -> workspace_memory = 0;
        // Insert the begin and end event.
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventRecord(start,0);

        size_t size = 1;
        for (index_t i = 0; i < input->ndim; i++) {
          size *= input->shape[i];
        }
        dim3 blocks;
        dim3 threads;
        float *output_data = (float *)output->data;
        const float *input_data = (const float *)input->data;
        if (size <= 1024) {
          threads.x = size;
          blocks.x = 1;
        } else {
          threads.x = 1024;
          blocks.x = (size + 1023) / 1024;
        }
        if (stream_handle)
          add_const_kernel<<<blocks, threads, 0, *(hipStream_t*)stream_handle->handle>>>(input_data, output_data, val, size);
        else
          add_const_kernel<<<blocks, threads>>>(input_data, output_data, val, size);

        float elapsedTime;
        hipEventCreate(&stop);
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start,stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        p->time = elapsedTime;
    }
    else{
      size_t size = 1;
      for (index_t i = 0; i < input->ndim; i++) {
        size *= input->shape[i];
      }
      dim3 blocks;
      dim3 threads;
      float *output_data = (float *)output->data;
      const float *input_data = (const float *)input->data;
      if (size <= 1024) {
        threads.x = size;
        blocks.x = 1;
      } else {
        threads.x = 1024;
        blocks.x = (size + 1023) / 1024;
      }
      if (stream_handle)
        add_const_kernel<<<blocks, threads, 0, *(hipStream_t*)stream_handle->handle>>>(input_data, output_data, val, size);
      else
        add_const_kernel<<<blocks, threads>>>(input_data, output_data, val, size);
    }
    return 0;
}