#include "hip/hip_runtime.h"
#include "gpu_runtime.h"

__global__ void ele_add_kernel(const float *matA, const float *matB, float *output, size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size) return;
    output[ind] = matA[ind] + matB[ind];
  }
  
  int DLGpuMatrixElementwiseAdd(const DLArrayHandle matA,
                                const DLArrayHandle matB, DLArrayHandle output, DLStreamHandle stream_handle = NULL, ProfilerHandle p = NULL){
  if(p != NULL){
    int size_a = 1, size_b = 1, size_c = 1;
    for(int i = 0; i < matA -> ndim; i++)
        size_a *= matA -> shape[i];
    for(int i = 0; i < matB -> ndim; i++)
        size_b *= matB -> shape[i];
    for(int i = 0; i < output -> ndim; i++)
        size_c *= output -> shape[i];
    p -> input_memory = 1.0 * (size_a  + size_b) * sizeof(float) / 1024 / 1024;
    p -> output_memory = 1.0 * size_c * sizeof(float) / 1024 / 1024;
    p -> workspace_memory = 0;
    // Insert the begin and end event.
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventRecord(start,0);

    int dev_id = (matA->ctx).device_id;
    hipSetDevice(dev_id);
    size_t size = 1;
    for (index_t i = 0; i< matA->ndim; i++) {
      size *= matA->shape[i];
    }
    dim3 blocks;
    dim3 threads;
    float *output_data = (float *)output->data;
    const float *matA_data = (const float *)matA->data;
    const float *matB_data = (const float *)matB->data;
    if (size <= 1024) {
      threads.x = size;
      blocks.x = 1;
    } else {
      threads.x = 1024;
      blocks.x = (size + 1023) / 1024;
    }
    if (stream_handle)
      ele_add_kernel<<<blocks, threads, 0, *(hipStream_t*)stream_handle->handle>>>(matA_data, matB_data, output_data, size);
    else
      ele_add_kernel<<<blocks, threads>>>(matA_data, matB_data, output_data, size);

    float elapsedTime;
    hipEventCreate(&stop);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start,stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    p->time = elapsedTime;
  }else{
    int dev_id = (matA->ctx).device_id;
    hipSetDevice(dev_id);
    size_t size = 1;
    for (index_t i = 0; i< matA->ndim; i++) {
      size *= matA->shape[i];
    }
    dim3 blocks;
    dim3 threads;
    float *output_data = (float *)output->data;
    const float *matA_data = (const float *)matA->data;
    const float *matB_data = (const float *)matB->data;
    if (size <= 1024) {
      threads.x = size;
      blocks.x = 1;
    } else {
      threads.x = 1024;
      blocks.x = (size + 1023) / 1024;
    }
    if (stream_handle)
      ele_add_kernel<<<blocks, threads, 0, *(hipStream_t*)stream_handle->handle>>>(matA_data, matB_data, output_data, size);
    else
      ele_add_kernel<<<blocks, threads>>>(matA_data, matB_data, output_data, size);
  }
    return 0;
}