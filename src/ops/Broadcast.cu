#include "hip/hip_runtime.h"
#include "gpu_runtime.h"


__global__ void broadcast_to_kernel(const float *input_data,float *output_data,size_t input_size,size_t output_size){
  size_t id = blockIdx.x * blockDim.x +threadIdx.x;
  if(id >= output_size)return ;
  output_data[id] = input_data[id%input_size];
}
  
int DLGpuBroadcastTo(const DLArrayHandle input, DLArrayHandle output, DLStreamHandle stream_handle = NULL, ProfilerHandle p = NULL){

  if(p != NULL){
      int size_input = 1, size_output = 1;
      for(int i = 0; i < input -> ndim; i++)
          size_input *= input -> shape[i];
      for(int i = 0; i < output -> ndim; i++)
          size_output *= output -> shape[i];
      p -> input_memory = 1.0 * (size_input) * sizeof(float) / 1024 / 1024;
      p -> output_memory = 1.0 * size_output * sizeof(float) / 1024 / 1024;
      p -> workspace_memory = 0;
    // Insert the begin and end event.
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventRecord(start,0);

    for(index_t i = 0; i < input->ndim; i++){
      assert((input->shape[i]) == (output->shape[i+1]));
    }
    size_t input_size = 1;
    for(index_t i = 0;i < input->ndim; i++){
      input_size *= input->shape[i];
    }
    size_t output_size = input_size * (output->shape[0]);
    size_t BLOCKS = (output_size + THREADS_PER_BLOCK - 1)/ THREADS_PER_BLOCK;
    if (stream_handle)
    {
      hipStream_t *s = (hipStream_t*)(stream_handle->handle);
      broadcast_to_kernel<<<BLOCKS,THREADS_PER_BLOCK, 0, *s>>>((const float*)(input->data),(float*)(output->data),input_size,output_size);
      //broadcast_to_kernel<<<BLOCKS,THREADS_PER_BLOCK, 0, *(hipStream_t*)stream_handle->handle>>>((const float*)(input->data),(float*)(output->data),input_size,output_size);
    }
    else
      broadcast_to_kernel<<<BLOCKS,THREADS_PER_BLOCK>>>((const float*)(input->data),(float*)(output->data),input_size,output_size);
  
    float elapsedTime;
    hipEventCreate(&stop);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start,stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    p->time = elapsedTime; 
  }else{
    for(index_t i = 0; i < input->ndim; i++){
      assert((input->shape[i]) == (output->shape[i+1]));
    }
    size_t input_size = 1;
    for(index_t i = 0;i < input->ndim; i++){
      input_size *= input->shape[i];
    }
    size_t output_size = input_size * (output->shape[0]);
    size_t BLOCKS = (output_size + THREADS_PER_BLOCK - 1)/ THREADS_PER_BLOCK;
    if (stream_handle)
    {
      hipStream_t *s = (hipStream_t*)(stream_handle->handle);
      broadcast_to_kernel<<<BLOCKS,THREADS_PER_BLOCK, 0, *s>>>((const float*)(input->data),(float*)(output->data),input_size,output_size);
      //broadcast_to_kernel<<<BLOCKS,THREADS_PER_BLOCK, 0, *(hipStream_t*)stream_handle->handle>>>((const float*)(input->data),(float*)(output->data),input_size,output_size);
    }
    else
      broadcast_to_kernel<<<BLOCKS,THREADS_PER_BLOCK>>>((const float*)(input->data),(float*)(output->data),input_size,output_size);  
  }
  return 0;
}
