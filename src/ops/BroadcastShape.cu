#include "hip/hip_runtime.h"
#include "gpu_runtime.h"


__global__ void broadcast_shape_kernel(
        const float *input_data, float *output_data, 
        uint* out_strides, uint* in_dims, 
        size_t ndims, size_t output_size){
    size_t o_ind = blockIdx.x * blockDim.x +threadIdx.x;
    if(o_ind >= output_size) return;
    size_t i_ind = 0;
    uint temp = o_ind;
    for (int i = 0; i < ndims; ++i) {
        i_ind *= in_dims[i];
        uint adder = temp / out_strides[i];
        if (in_dims[i] > 1) {
            i_ind += adder;
        }
        temp %= out_strides[i];
    }
    output_data[o_ind] = input_data[i_ind];
}

int DLGpuBroadcastShape(const DLArrayHandle in_arr, DLArrayHandle out_arr, int* add_axes, DLStreamHandle stream_handle = NULL) {
    size_t allocated = out_arr->ndim * sizeof(uint);
    uint* out_strides = (uint*)malloc(allocated);
    uint* in_dims = (uint*)malloc(allocated);
    // uint *out_strides;
    // uint *in_dims;
    // CUDA_CALL(hipHostMalloc((void**)&out_strides, allocated));
    // CUDA_CALL(hipHostMalloc((void**)&in_dims, allocated));
    size_t output_size = 1;
    size_t diff = out_arr->ndim - in_arr->ndim;

    if (add_axes == NULL) {
        for (int i = out_arr->ndim - 1; i >= 0; --i) {
            out_strides[i] = output_size;
            output_size *= out_arr->shape[i];
            if (i < diff) {
                in_dims[i] = 1;
            } else {
                in_dims[i] = in_arr->shape[i-diff];
            }
        }        
    } else {
        for (int i = out_arr->ndim - 1; i >= 0; --i) {
            out_strides[i] = output_size;
            output_size *= out_arr->shape[i];
            in_dims[i] = 0;
        }
        for (int i = 0; i < diff; ++i) {
            in_dims[add_axes[i]] = 1;
        }
        int o_ind = 0;
        for (int i = 0; i < in_arr->ndim; ++i) {
            while (in_dims[o_ind++] == 1);
            in_dims[o_ind-1] = in_arr->shape[i];
        }
    }
    int dev_id = (in_arr->ctx).device_id;
    if (is_chunk_init(dev_id) == false) {
        chunk_init(dev_id);
    }
    // uint* gpu_strides = NULL;
    // uint*gpu_dims = NULL;
    // std::cout<<"broadcast: "<<allocated<<std::endl;
    // // std::cout<<"device id"<<dev_id<<"  broadcast"<<std::endl;
    // CUDA_CALL(hipSetDevice(dev_id));
    // CUDA_CALL(hipMalloc((void**)&gpu_strides, allocated));
    // CUDA_CALL(hipMalloc((void**)&gpu_dims, allocated));
    // std::cout<<"ok"<<std::endl;
    // uint* gpu_strides = (uint*) MemoryPool -> DLMemoryMalloc(allocated);
    // uint* gpu_dims = (uint*) MemoryPool -> DLMemoryMalloc(allocated);
    uint* gpu_strides = (uint*)find_chunk(allocated, dev_id);
    uint* gpu_dims = (uint*)find_chunk(allocated, dev_id);

    CUDA_CALL(hipMemcpy(gpu_strides, out_strides, allocated, hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(gpu_dims, in_dims, allocated, hipMemcpyHostToDevice));
    // std::cout<<gpu_strides<<" "<<out_strides<<" "<<allocated<<std::endl;
    // CUDA_CALL(hipMemcpyAsync(gpu_strides, out_strides, allocated, hipMemcpyHostToDevice, *(hipStream_t*)stream_handle->handle));
    // CUDA_CALL(hipMemcpyAsync(gpu_dims, in_dims, allocated, hipMemcpyHostToDevice, *(hipStream_t*)stream_handle->handle));
    
    dim3 blocks;
    dim3 threads;
    if (output_size <= 1024) {
        threads.x = output_size;
        blocks.x = 1;
    } else {
        threads.x = 1024;
        blocks.x = (output_size + 1023) / 1024;
    }
    if (stream_handle) {
        hipStream_t *s = (hipStream_t*)(stream_handle->handle);
        broadcast_shape_kernel<<<blocks, threads, 0, *s>>>(
            (const float*)(in_arr->data), (float*)(out_arr->data), gpu_strides, gpu_dims, (size_t)out_arr->ndim, output_size);
    } else {
        broadcast_shape_kernel<<<blocks, threads>>>(
            (const float*)(in_arr->data), (float*)(out_arr->data), gpu_strides, gpu_dims, (size_t)out_arr->ndim, output_size);
    }
    // MemoryPool -> DLMemoryFree(gpu_strides, allocated);
    // MemoryPool -> DLMemoryFree(gpu_dims, allocated);
    del_chunk(gpu_strides, dev_id);
    del_chunk(gpu_dims, dev_id);
    // CUDA_CALL(hipFree(gpu_strides));
    // CUDA_CALL(hipFree(gpu_dims));
    free(out_strides);
    free(in_dims);
    return 0;
}

// #include "gpu_runtime.h"

// __global__ void broadcast_shape_kernel(
//         const float *input_data, float *output_data, 
//         int input_size, int output_size){
//     size_t o_ind = blockIdx.x * blockDim.x +threadIdx.x;
//     if(o_ind >= output_size) return;
//     output_data[o_ind] = input_data[o_ind % input_size];
// }

// int DLGpuBroadcastShape(const DLArrayHandle in_arr, DLArrayHandle out_arr, int* add_axes, DLStreamHandle stream_handle = NULL) {
//     size_t input_size = 1;
//     size_t output_size = 1;
//     for(int i = 0; i < in_arr->ndim; i++){
//         input_size *= in_arr->shape[i];
//     }
//     for(int i = 0; i < out_arr->ndim; i++){
//         output_size *= out_arr->shape[i];
//     }
//     dim3 blocks;
//     dim3 threads;
//     if (output_size <= 1024) {
//         threads.x = output_size;
//         blocks.x = 1;
//     } else {
//         threads.x = 1024;
//         blocks.x = (output_size + 1023) / 1024;
//     }
//     if (stream_handle) {
//         hipStream_t *s = (hipStream_t*)(stream_handle->handle);
//         broadcast_shape_kernel<<<blocks, threads, 0, *s>>>(
//             (const float*)(in_arr->data), (float*)(out_arr->data), input_size, output_size);
//     } else {
//         broadcast_shape_kernel<<<blocks, threads>>>(
//             (const float*)(in_arr->data), (float*)(out_arr->data), input_size, output_size);
//     }
//     return 0;
// }