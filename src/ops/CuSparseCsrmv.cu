#include "gpu_runtime.h"

int CuSparse_DLGpuCsrmv(const DLArrayHandle data_handle,
                   const DLArrayHandle row_handle,
                   const DLArrayHandle col_handle,
                   int nrow, int ncol,
                   bool transpose,
                   const DLArrayHandle input_handle,
                   DLArrayHandle output_handle, DLStreamHandle stream_handle = NULL, ProfilerHandle p = NULL){

  assert (data_handle->ndim == 1);
  /*
  using namespace std;
  std::cout << "Data_hadle_dim:" << data_handle->ndim << std::endl;
  std::cout << "Data_hadle_shape[0]:" << data_handle->shape[0] << std::endl;

  std::cout << "Row_hadle_dim:" << row_handle->ndim << std::endl;
  std::cout << "Row_hadle_shape[0]:" << row_handle->shape[0] << std::endl;

  std::cout << "Col_hadle_dim:" << col_handle->ndim << std::endl;
  std::cout << "Col_hadle_shape[0]:" << col_handle->shape[0] << std::endl;

  std::cout << "nrow:" << nrow << std::endl;
  std::cout << "ncol:" << ncol << std::endl;
  */
  assert (row_handle->ndim == 1);
  assert (col_handle->ndim == 1);
  assert (transpose ? nrow == input_handle->shape[0] : ncol == input_handle->shape[0]);

  int nnz = data_handle->shape[0];
  int dev_id = (data_handle->ctx).device_id;
  cusp_init(dev_id, stream_handle);


  float alpha = 1.0;
  float beta = 0.0;

  hipsparseMatDescr_t descr = 0;
  CUSP_CALL(hipsparseCreateMatDescr(&descr));
  hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseOperation_t trans = transpose ? HIPSPARSE_OPERATION_TRANSPOSE : HIPSPARSE_OPERATION_NON_TRANSPOSE;
  CUSP_CALL(hipsparseScsrmv(cusp_map[dev_id], trans,
         nrow, ncol, nnz, (const float*)&alpha, 
         descr, (const float*)data_handle->data, 
         (const int*)row_handle->data, (const int*)col_handle->data, (const float*)input_handle->data,
         (const float*)&beta, (float*)output_handle->data));
  if(p != NULL){
    int size_input1 = 1, size_input2 = 1, size_input3 = 1, size_input4 = 1, size_output = 1;
    for(int i = 0; i < data_handle -> ndim; i++)
        size_input1 *= data_handle -> shape[i];
    for(int i = 0; i < row_handle -> ndim; i++)
        size_input2 *= row_handle -> shape[i];
    for(int i = 0; i < col_handle -> ndim; i++)
        size_input3 *= col_handle -> shape[i];
    for(int i = 0; i < input_handle -> ndim; i++)
        size_input4 *= input_handle -> shape[i];
    for(int i = 0; i < output_handle -> ndim; i++)
        size_output *= output_handle -> shape[i];
    p -> input_memory = 1.0 * (size_input1 + size_input2 + size_input3) * sizeof(float) / 1024 / 1024;
    p -> output_memory = 1.0 * size_output * sizeof(float) / 1024 / 1024;
    p -> workspace_memory = 0;
  }
  return 0;
}