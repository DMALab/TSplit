#include "gpu_runtime.h"

int CuDNN_DLGpuAvgerage_Pooling2d(const DLArrayHandle input,const size_t kernel_H, const size_t kernel_W, DLArrayHandle output, const size_t padding, const size_t stride, DLStreamHandle stream_handle = NULL, ProfilerHandle p = NULL){
    // create handle
    // CUDNN_CALL(hipdnnCreate(&cudnn));
    int dev_id = (input->ctx).device_id;
    cudnn_init(dev_id, stream_handle);
    
    // input
    size_t input_N = input->shape[0];
    size_t input_C = input->shape[1];
    size_t input_H = input->shape[2];
    size_t input_W = input->shape[3];
    const float * input_data = (const float*) input->data;
    
    //output
    size_t output_H = output->shape[2];
    size_t output_W = output->shape[3];
    float *output_data = (float *) output->data;
  
    //pooling descriptor
    hipdnnPoolingDescriptor_t avgpool_desc;
    CUDNN_CALL(hipdnnCreatePoolingDescriptor(&avgpool_desc));
    // std::cout<<"padding = "<<padding<<" stride = "<<stride<<std::endl;
    CUDNN_CALL(hipdnnSetPooling2dDescriptor(avgpool_desc,HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING,
               HIPDNN_PROPAGATE_NAN, kernel_H, kernel_W, padding, padding, stride, stride));
    
    //input descriptor
    hipdnnTensorDescriptor_t input_desc;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_desc));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, input_N, input_C, input_H, input_W));
  
    //output descriptor 
    hipdnnTensorDescriptor_t output_desc;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_desc));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(output_desc,HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, input_N, input_C, output_H, output_W));
  
    float alpha = 1.0f;
    float beta = 0.0f;
  
    CUDNN_CALL(hipdnnPoolingForward(cudnn_map[dev_id], avgpool_desc, 
                                  &alpha, input_desc, input_data, &beta, output_desc, output_data));
  
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_desc));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_desc));
    CUDNN_CALL(hipdnnDestroyPoolingDescriptor(avgpool_desc));
    // CUDNN_CALL(hipdnnDestroy(cudnn));
    if(p != NULL){
        int size_input = 1, size_output = 1;
        for(int i = 0; i < input -> ndim; i++)
            size_input *= input -> shape[i];
        for(int i = 0; i < output -> ndim; i++)
            size_output *= output -> shape[i];
        p -> input_memory = 1.0 * (size_input) * sizeof(float) / 1024 / 1024;
        p -> output_memory = 1.0 * size_output * sizeof(float) / 1024 / 1024;
        p -> workspace_memory = 0;
    }
    return 0;
  }
  
  int CuDNN_DLGpuAvgerage_Pooling2d_gradient(const DLArrayHandle output_Y,const DLArrayHandle gradient_Y,const DLArrayHandle input_X, const size_t kernel_H, const size_t kernel_W, DLArrayHandle gradient_X, const size_t padding, const size_t stride, DLStreamHandle stream_handle = NULL, ProfilerHandle p = NULL){
    // create handle
    // CUDNN_CALL(hipdnnCreate(&cudnn));
    int dev_id = (input_X->ctx).device_id;
    cudnn_init(dev_id, stream_handle);
      
    // input
    size_t input_N = input_X->shape[0];
    size_t input_C = input_X->shape[1];
    size_t input_H = input_X->shape[2];
    size_t input_W = input_X->shape[3];
    const float * input_data = (const float*) input_X->data;
    float * gradient_x_data = (float *) gradient_X->data; 
    //output
    size_t output_H = output_Y->shape[2];
    size_t output_W = output_Y->shape[3];
    const float *output_data = (const float *) output_Y->data;
    const float *gradient_Y_data = (const float *) gradient_Y -> data;
  
    //pooling descriptor
    hipdnnPoolingDescriptor_t avgpool_desc;
    CUDNN_CALL(hipdnnCreatePoolingDescriptor(&avgpool_desc));
    // std::cout<<"padding = "<<padding<<" stride = "<<stride<<std::endl;
    CUDNN_CALL(hipdnnSetPooling2dDescriptor(avgpool_desc,HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING,
              HIPDNN_PROPAGATE_NAN, kernel_H, kernel_W, padding, padding, stride, stride));
  
    //input descriptor
    hipdnnTensorDescriptor_t input_desc;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_desc));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, input_N, input_C, input_H, input_W));
  
    //output descriptor 
    hipdnnTensorDescriptor_t output_desc;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_desc));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(output_desc,HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, input_N, input_C, output_H, output_W));
  
    float alpha = 1.0f;
    float beta = 0.0f;
  
    CUDNN_CALL(hipdnnPoolingBackward(cudnn_map[dev_id],  avgpool_desc,
                                    &alpha, output_desc, output_data,
                                    output_desc, gradient_Y_data,
                                    input_desc, input_data,
                                    &beta, input_desc,gradient_x_data));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_desc));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_desc));
    CUDNN_CALL(hipdnnDestroyPoolingDescriptor(avgpool_desc));
    // CUDNN_CALL(hipdnnDestroy(cudnn));

  if(p != NULL){
    int size_a = 1, size_b = 1, size_c = 1, size_d = 1;
    for(int i = 0; i < output_Y -> ndim; i++)
        size_a *= output_Y -> shape[i];
    for(int i = 0; i < gradient_Y -> ndim; i++)
        size_b *= gradient_Y -> shape[i];
    for(int i = 0; i < gradient_X -> ndim; i++)
        size_c *= gradient_X -> shape[i];
    for(int i = 0; i < input_X -> ndim; i++)
        size_d *= input_X -> shape[i];
    p -> input_memory = 1.0 * (size_a  + size_b + size_d) * sizeof(float) / 1024 / 1024;
    p -> output_memory = 1.0 * size_c * sizeof(float) / 1024 / 1024;
    p -> workspace_memory = 0;
  }
    return 0;
}
  