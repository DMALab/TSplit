#include "hip/hip_runtime.h"
#include "gpu_runtime.h"

__global__ void ele_div_kernel(const float *matA, const float *matB, float *output, size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size) return;
    output[ind] = matA[ind] / matB[ind];
  }
  
int DLGpuMatrixElementwiseDivide(const DLArrayHandle matA,
                                   const DLArrayHandle matB,
                                   DLArrayHandle output, DLStreamHandle stream_handle = NULL, ProfilerHandle p = NULL){
    /* TODO: Your code here */
    size_t size = 1;
    for (index_t i = 0; i < matA->ndim; i++) {
      size *= matA->shape[i];
    }
    dim3 blocks;
    dim3 threads;
    float *output_data = (float *)output->data;
    const float *matA_data = (const float *)matA->data;
    const float *matB_data = (const float *)matB->data;
    if (size <= 1024) {
      threads.x = size;
      blocks.x = 1;
    } else {
      threads.x = 1024;
      blocks.x = (size + 1023) / 1024;
    }
    if (stream_handle)
      ele_div_kernel<<<blocks, threads, 0, *(hipStream_t*)stream_handle->handle>>>(matA_data, matB_data, output_data, size);
    else
      ele_div_kernel<<<blocks, threads>>>(matA_data, matB_data, output_data, size);

  if(p != NULL){
    int size_a = 1, size_b = 1, size_c = 1;
    for(int i = 0; i < matA -> ndim; i++)
        size_a *= matA -> shape[i];
    for(int i = 0; i < matB -> ndim; i++)
        size_b *= matB -> shape[i];
    for(int i = 0; i < output -> ndim; i++)
        size_c *= output -> shape[i];
    p -> input_memory = 1.0 * (size_a  + size_b) * sizeof(float) / 1024 / 1024;
    p -> output_memory = 1.0 * size_c * sizeof(float) / 1024 / 1024;
    p -> workspace_memory = 0;
  }
    return 0;
  }