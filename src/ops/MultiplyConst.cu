#include "hip/hip_runtime.h"
#include "gpu_runtime.h"

__global__ void mult_const_kernel(const float *input, float *output, float value, size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size) return;
    output[ind] = input[ind] * value;
}
  
int DLGpuMatrixMultiplyByConst(const DLArrayHandle input, float val,
                                 DLArrayHandle output, DLStreamHandle stream_handle = NULL, ProfilerHandle p = NULL){
    /* TODO: Your code here */
    int dev_id = (input->ctx).device_id;
    hipSetDevice(dev_id);
    // printf("dev_id = %d\n",dev_id);
    size_t size = 1;
    for (index_t i = 0; i < input->ndim; i++) {
      size *= input->shape[i];
    }
    dim3 blocks;
    dim3 threads;
    float *output_data = (float *)output->data;
    const float *input_data = (const float *)input->data;
    if (size <= 1024) {
      threads.x = size;
      blocks.x = 1;
    } else {
      threads.x = 1024;
      blocks.x = (size + 1023) / 1024;
    }
    if (stream_handle)
      mult_const_kernel<<<blocks, threads, 0, *(hipStream_t*)stream_handle->handle>>>(input_data, output_data, val, size);
    else
      mult_const_kernel<<<blocks, threads>>>(input_data, output_data, val, size);
    if(p != NULL){
      int size_input = 1, size_output = 1;
      for(int i = 0; i < input -> ndim; i++)
          size_input *= input -> shape[i];
      for(int i = 0; i < output -> ndim; i++)
          size_output *= output -> shape[i];
      p -> input_memory = 1.0 * (size_input) * sizeof(float) / 1024 / 1024;
      p -> output_memory = 1.0 * size_output * sizeof(float) / 1024 / 1024;
      p -> workspace_memory = 0;
    }
    return 0;
}