#include "hip/hip_runtime.h"
#include "gpu_runtime.h"

extern __global__ void float_memory_copy(float *A, const float *B, size_t len);

int DLGpuReshape(const DLArrayHandle in_arr, DLArrayHandle out_arr, DLStreamHandle stream_handle = NULL, ProfilerHandle p = NULL){
  size_t input_size = 1;
  size_t output_size = 1;
  for(index_t i = 0; i < in_arr -> ndim; i++){
    input_size *= in_arr ->shape[i];
  }
  for(index_t i = 0; i < out_arr -> ndim; i++){
    output_size *= out_arr ->shape[i];
  }
  // printf("input = %d, output = %d\n",input_size, output_size);
  assert(input_size == output_size);
  const float * input_data = (const float *) in_arr->data;
  float *output_data = (float *) out_arr->data;
  size_t BLOCKS = (input_size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  if (stream_handle)
    float_memory_copy<<<BLOCKS, THREADS_PER_BLOCK, 0, *(hipStream_t*)stream_handle->handle>>>(output_data, input_data, input_size);
  else
    float_memory_copy<<<BLOCKS, THREADS_PER_BLOCK>>>(output_data, input_data, input_size);
  if(p != NULL){
    int size_input = 1, size_output = 1;
    for(int i = 0; i < in_arr -> ndim; i++)
        size_input *= in_arr -> shape[i];
    for(int i = 0; i < out_arr -> ndim; i++)
        size_output *= out_arr -> shape[i];
    p -> input_memory = 1.0 * (size_input) * sizeof(float) / 1024 / 1024;
    p -> output_memory = 1.0 * size_output * sizeof(float) / 1024 / 1024;
    p -> workspace_memory = 0;
  }
  return 0;
}