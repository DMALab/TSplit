#include "hip/hip_runtime.h"
#include "gpu_runtime.h"

__global__ void sqrt_kernel(float *input, float *output, size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size) return;
    output[ind] = sqrtf(input[ind]);
  }
  
int DLGpuSqrt(const DLArrayHandle input, DLArrayHandle output, DLStreamHandle stream_handle = NULL, ProfilerHandle p = NULL){
    /* TODO: Your code here */
    size_t size = 1;
    for (index_t i = 0; i < input->ndim; i++) {
      size *= input->shape[i];
    }
    dim3 blocks;
    dim3 threads;
    float *input_data = (float *)input->data;
    float *output_data = (float *)output->data;
    if (size <= 1024) {
      threads.x = size;
      blocks.x = 1;
    } else {
      threads.x = 1024;
      blocks.x = (size + 1023) / 1024;
    }
    if (stream_handle)
      sqrt_kernel<<<blocks, threads, 0, *(hipStream_t*)stream_handle->handle>>>(input_data, output_data, size);
    else
      sqrt_kernel<<<blocks, threads>>>(input_data, output_data, size);
    if(p != NULL){
      int size_input = 1, size_output = 1;
      for(int i = 0; i < input -> ndim; i++)
          size_input *= input -> shape[i];
      for(int i = 0; i < output -> ndim; i++)
          size_output *= output -> shape[i];
      p -> input_memory = 1.0 * (size_input) * sizeof(float) / 1024 / 1024;
      p -> output_memory = 1.0 * size_output * sizeof(float) / 1024 / 1024;
      p -> workspace_memory = 0;
    }
    return 0;
}

__global__ void reciprocal_sqrt_kernel(float *input, float *output, size_t size) {
  size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
  if (ind >= size) return;
  output[ind] = rsqrtf(input[ind]);
}

int DLGpuReciprocalSqrt(const DLArrayHandle input, DLArrayHandle output, DLStreamHandle stream_handle = NULL, ProfilerHandle p = NULL){
  /* TODO: Your code here */
  size_t size = 1;
  for (index_t i = 0; i < input->ndim; i++) {
    size *= input->shape[i];
  }
  dim3 blocks;
  dim3 threads;
  float *input_data = (float *)input->data;
  float *output_data = (float *)output->data;
  if (size <= 1024) {
    threads.x = size;
    blocks.x = 1;
  } else {
    threads.x = 1024;
    blocks.x = (size + 1023) / 1024;
  }
  if (stream_handle)
    reciprocal_sqrt_kernel<<<blocks, threads, 0, *(hipStream_t*)stream_handle->handle>>>(input_data, output_data, size);
  else
    reciprocal_sqrt_kernel<<<blocks, threads>>>(input_data, output_data, size);

    if(p != NULL){
      int size_input = 1, size_output = 1;
      for(int i = 0; i < input -> ndim; i++)
          size_input *= input -> shape[i];
      for(int i = 0; i < output -> ndim; i++)
          size_output *= output -> shape[i];
      p -> input_memory = 1.0 * (size_input) * sizeof(float) / 1024 / 1024;
      p -> output_memory = 1.0 * size_output * sizeof(float) / 1024 / 1024;
      p -> workspace_memory = 0;
    }
  return 0;
}